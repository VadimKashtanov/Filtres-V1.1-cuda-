#include "main.cuh"

PAS_OPTIMISER()
static void titre_partie(char * str) {
	printf("========= %s =========\n", str);
};

PAS_OPTIMISER()
int main(int argc, char ** argv) {
	//	-- Init --
	srand(0);
	hipSetDevice(0);

	titre_partie("Charger tout"); charger_tout();
    //titre_partie("   Verif f  ");      verif_f();
	titre_partie("Performances"); performances();
	titre_partie("  Verif df  ");     verif_df();

	//===============
	titre_partie("  Programme Generale  ");

	//	-- Fin --
	liberer_tout();
};
