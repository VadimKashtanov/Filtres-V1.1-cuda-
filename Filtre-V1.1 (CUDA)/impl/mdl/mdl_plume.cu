#include "hip/hip_runtime.h"
#include "mdl.cuh"

void taille_mdl(Mdl_t * mdl) {
	printf("  sizeof(Mdl_t) ~= %3.3f Mo\n",
		(float)sizeof(float) * (mdl->FILTRES + mdl->POIDS + mdl->VARS + mdl->LOCDS) / 1e6
	);
};

void plume_mdl(Mdl_t * mdl) {
	printf("Mdl_t Filtres=%i C=%i (VARS=%i FILTRES=%i POIDS=%i LOCDS=%i)\n",
		mdl->FS, mdl->C,
		mdl->VARS, mdl->FILTRES, mdl->POIDS, mdl->LOCDS);
	printf(" 0| filtre [%3.i] DEPART_VARS=%i\n",
		mdl->FS,
		mdl->DEPART_VARS[0]
	);
	FOR(0, i, EMA_INTS)
		if (mdl->F_par_EMA_INT[i] != 0)
			printf("\t (%2.i) %4.i filtres en ema=%i intervalle=%i\n", ema_ints[i].ligne, mdl->F_par_EMA_INT[i], ema_ints[i].ema, ema_ints[i].interv);
	printf("EMA_INT_BLOQUE_ligne : "); FOR(0, i, mdl->BLOQUES) {printf("%i ", mdl->EMA_INT_BLOQUE_ligne[i]);};printf("\n");
	FOR(1, i, mdl->C) {
		printf("%2.i| pensee:dot1d(tanh) [%4.i]   DEPART_VARS=%i DEPART_POIDS=%i DEPART_LOCDS=%i  (poids=%i)\n",
			i,
			mdl->ST[i],
			mdl->DEPART_VARS[i],
			mdl->DEPART_POIDS[i],
			mdl->DEPART_LOCDS[i],

			(mdl->ST[i-1]+1)*mdl->ST[i]
		);
	}
};