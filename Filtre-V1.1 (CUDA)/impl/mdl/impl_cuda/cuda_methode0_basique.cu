#include "hip/hip_runtime.h"
#include "mdl.cuh"

/*	Filtres : Memoire constante
	Poids   : Memoire constante */

__device__ static float filtre(float * x, float * dif_x, float * f, float * dif_f) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	return 2*expf(-s*s -d*d)-1;
};

__device__ static float perceptron(float * x, float * p, uint _N) {
	float s = p[_N-1+1];
	FOR(0, i, _N) s += x[i]*p[i];
	return ACTIV(s);
};

__global__ void kerd_mdl(
	uint C,
	uint * EMA_INT_BLOQUE_ligne__d,
	uint  BLOQUES,
	uint * ST__d,
	//
	uint * DEPART_POIDS__d,
	float * f__d, float * p__d,
	float * dif_f__d,
	uint t0, uint t1, float * res__d,
	float * normalisee__d, float * dif_normalisee__d)
{
	uint t = t0 + (threadIdx.x + blockIdx.x * blockDim.x);
	//
	if (t < t1) {
		float r0[MAX_Y];
		float r1[MAX_Y];

		//	------------------------------------

		FOR(0, b, BLOQUES) {
			FOR(0, i, BLOQUE_ST) {
				uint ligne = EMA_INT_BLOQUE_ligne__d[b];
				r0[b*BLOQUE_ST + i] = filtre(
						normalisee__d + ligne*PRIXS*N_FLTR + t*N_FLTR,
					dif_normalisee__d + ligne*PRIXS*N_FLTR + t*N_FLTR,
					f__d     + b*BLOQUE_ST*N     + i*N,
					dif_f__d + b*BLOQUE_ST*(N-1) + i*(N-1)
				);
			//	printf("(%i)%f\n", b*BLOQUE_ST + i, r0[b*BLOQUE_ST + i]);

			};
		};
		
		//	Pensee
		FOR(1, c, C) {
			//printf("==========\n");
			FOR(0, y, ST__d[c]) {
				(c%2==0 ? r0 : r1)[y] = perceptron(
					(c%2==0 ? r1 : r0),
					p__d + DEPART_POIDS__d[c] + y*(ST__d[c-1]+1),
					ST__d[c-1]
				);
				//printf("(%i) %+f (%f)\n", y, (c%2==0 ? r0 : r1)[y], *(p__d + DEPART_POIDS__d[c] + y*(ST__d[c-1]+1) + ST__d[c-1]));
			};
		};

		res__d[t-t0] = ((C-1)%2==0 ? r0 : r1)[0];
	};
};

void cuda_mdt0_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));

	//	--- Mdl_t ---
	kerd_mdl<<<dim3(KER_DIV((t1-t0), 256)), dim3(256)>>>(
		mdl->C,
		mdl->EMA_INT_BLOQUE_ligne__d,
		mdl->BLOQUES,
		mdl->ST__d,
		//
		mdl->DEPART_POIDS__d,
		mdl->f_d,
		mdl->p_d,
		mdl->dif_f_d,
		t0, t1, res_d,
		normalisee__d, dif_normalisee__d
	);
	ATTENDRE_KER_CUDA();

	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));

	//
	CONTROLE_CUDA(hipFree(res_d));
};