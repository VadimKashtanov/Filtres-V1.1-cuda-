#include "hip/hip_runtime.h"
#include "mdl.cuh"

/*
	Faire une version avec une somme __partagée__
	et donc (BLOQUE_ST, N_FLT) avec BLOQUE_ST*N_FLT==256

*/

__global__ static
void filtre__kerd(
	uint t0,
	const uint * __restrict__ EMA_INT_BLOQUE_ligne__d,
	const float * __restrict__ normalisee__d,
	const float * __restrict__ f, const float * __restrict__ dif_f,
	float * __restrict__ y, uint Y_MAX)
{
	//	<<<(T, BLOQUES), (BLOQUE_ST)>>> //BLOQUE_ST*N_FLT==256

#define bloque blockIdx.y
#define f_dans_bloque threadIdx.x

	uint t = blockIdx.x;

	//	__partage__
	__shared__ uint ligne;
	if (threadIdx.x == 0)
		ligne = EMA_INT_BLOQUE_ligne__d[bloque];

	//	__partage__
	__shared__ float sh_x[N_FLTR];
	if (f_dans_bloque < N_FLTR)
		sh_x[f_dans_bloque] = normalisee__d[ligne*PRIXS*N_FLTR + (t0+t)*N_FLTR + f_dans_bloque];

	//	fonction de filtre
	float s = 0, d = 0;
	FOR(0, i, N_FLTR-1) {
		s += sqrtf(1 + fabs(     sh_x[i]        -   f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + i]  ));
		d += powf((1 + fabs((sh_x[i+1]-sh_x[i]) - dif_f[bloque*BLOQUE_ST*(N_FLTR-1) + f_dans_bloque*(N_FLTR-1) + i])), 2);
	};
	s += sqrtf(1 + fabs(sh_x[N_FLTR-1] - f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + N_FLTR-1]));

	//	s = a + b + c ...

	s = s/8-1;
	d = d/7-1;

	y[t*Y_MAX + bloque*BLOQUE_ST + f_dans_bloque] = 2*expf(-s*s -d*d)-1;
};

//=================================================

#define K__dot1d__X 16
#define K__dot1d__T  4

__global__ static
void dot1d__kerd(
	uint X,
	uint T, uint Y,
	uint Y_MAX,
	//
	uint c,
	const uint * DEPART_POIDS__d,
	//
	const float * __restrict__ x__d, const float * __restrict__ p__d, float * __restrict__ y__d)
{
	//	<<<(KERD(X,16),KERD(T,4)), (16,4)>>>
	const uint y = threadIdx.x + blockIdx.x * blockDim.x;
	const uint t = threadIdx.y + blockIdx.y * blockDim.y;

	if ( (t<T) && (y<Y) ) {
		float s = p__d[DEPART_POIDS__d[c] + y*(X+1) + X-1+1];
		FOR(0, i, X) {
			s += x__d[t*Y_MAX + i]*p__d[DEPART_POIDS__d[c] + y*(X+1) + i];
		}
		y__d[t*Y_MAX + y] = ACTIV(s);
	};
}

#define BLOCK_SIZE 32

__global__ void matvec_kernel(
	const float * __restrict__ dA,
	const float * __restrict__ dx,
	float * __restrict__ dy,
	const uint nRows,
	const uint nCols)
{
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ float x_shared[BLOCK_SIZE];

    T y_val = 0.0;

    #pragma unroll
    for (unsigned int m = 0; m < ((nCols + BLOCK_SIZE - 1)/ BLOCK_SIZE); ++m)
    {
        if ((m * BLOCK_SIZE + threadIdx.x) <  nCols) x_shared[threadIdx.x] = dx[threadIdx.x + m * BLOCK_SIZE];
        else                                         x_shared[threadIdx.x] = 0.f;
        __syncthreads();

        #pragma unroll
        for (unsigned int e = 0; e < BLOCK_SIZE; ++e) {
            // --- Column-major ordering - faster
            y_val += dA[tid + (e + BLOCK_SIZE * m) * nRows] * x_shared[e];
            // --- Row-major ordering - slower
            //y_val += dA[tid * nCols + (e + BLOCK_SIZE * m)] * x_shared[e];
        }

        __syncthreads();
    }

    if (tid < nRows) dy[tid] = y_val;

}

/*
__global__ static
void dot1d__mini_somme__kerd(

	)
{
	//	<<<(), ()>>>
	//	<<<(KERD(Y,16),KERD(T,8),KERD(X,16)), (16,8,8)>>>

	//	<<<(16Y, 8X, )>>>
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint t = threadIdx.y + blockIdx.y * blockDim.y;

	__shared__ float X16[16];
	if ()

	if ( (t<T) && (y<Y) ) {
		float s = p__d[DEPART_POIDS__d[c] + y*(X+1) + X-1+1];
		FOR(0, i, X) {
			s += x__d[t*Y_MAX + i]*p__d[DEPART_POIDS__d[c] + y*(X+1) + i];
		}
		y__d[t*Y_MAX + y] = ACTIV(s);
	};
};

__global__ static
void dot1d__ACTIV__kerd(
	)
{
	//	<<<(KERD(T, 32), KERD(X, 8)),  (32,8)>>>
	//	<<<(KERD(X,16),KERD(T,4)), (16,4)>>>
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint t = threadIdx.y + blockIdx.y * blockDim.y;

	if ( (t<T) && (y<Y) ) {
		y__d[t*Y_MAX + y] = ACTIV(y__d[t*Y_MAX + y]);
	};
};*/

//	==============================================================

static __global__ void enregistrer_les_resultats__kerd(
	float * res_d, float * y__d,
	uint Y_MAX, uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;

	if (t < T) {
		res_d[t] = y__d[t*Y_MAX + 0];
	};
};

void cuda_mdt2_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));

	//	--- Allocation des r0, r1 ---
	uint T = t1 - t0;
	float * r0__d, * r1__d;
	CONTROLE_CUDA(hipMalloc((void**)&r0__d, sizeof(float) * MAX_Y * T));
	CONTROLE_CUDA(hipMalloc((void**)&r1__d, sizeof(float) * MAX_Y * T));

	//	--- Mdl_t ---

	//	--- Filtres ---
	filtre__kerd<<<dim3(T, mdl->BLOQUES), dim3(BLOQUE_ST)>>>(
		t0,
		mdl->EMA_INT_BLOQUE_ligne__d,
		normalisee__d,
		mdl->f_d, mdl->dif_f_d,
		r0__d, MAX_Y
	);
	/*filtre__mini_kerd__kerd<<<dim3(T, mdl->BLOQUES), dim3(BLOQUE_ST, N_FLTR)>>>(
		t0,
		mdl->EMA_INT_BLOQUE_ligne__d,
		normalisee__d,
		mdl->f_d, mdl->dif_f_d,
		r0__d, MAX_Y
	);*/
	/*filtre__mini_kerd__kerd____v_s_puis_d<<<dim3(T, mdl->BLOQUES), dim3(BLOQUE_ST, N_FLTR)>>>(
		t0,
		mdl->EMA_INT_BLOQUE_ligne__d,
		normalisee__d,
		mdl->f_d, mdl->dif_f_d,
		r0__d, MAX_Y
	);*/
	ATTENDRE_KER_CUDA();

	//	--- Pensee Perceptronnale ---
	FOR(1, c, mdl->C) {
		dim3 grille(KER_DIV(mdl->ST[c], K__dot1d__X), KER_DIV(T, K__dot1d__T));
		dim3 noyaux(K__dot1d__X, K__dot1d__T);
		dot1d__kerd<<<grille, noyaux>>>(
			mdl->ST[c-1],
			T, mdl->ST[c],
			MAX_Y,
			//
			c,
			mdl->DEPART_POIDS__d,
			//
			(c%2==0 ? r1__d : r0__d),	//x__d
			mdl->p_d,
			(c%2==0 ? r0__d : r1__d)	//y__d
		);
		ATTENDRE_KER_CUDA();
	};

	//	--- Ecrire dans res ---
	enregistrer_les_resultats__kerd<<<dim3(KER_DIV(T,256)), dim3(256)>>>(
		res_d, ((mdl->C-1)%2==0 ? r0__d : r1__d),
		MAX_Y, T);
	ATTENDRE_KER_CUDA();
	
	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));

	//
	CONTROLE_CUDA(hipFree(res_d));
};