#include "hip/hip_runtime.h"
#include "mdl.cuh"

/*
	Faire une version avec une somme __partagée__
	et donc (BLOQUE_ST, N_FLT) avec BLOQUE_ST*N_FLT==256

*/

__global__ static
void filtre__kerd(
	uint t0,
	uint * EMA_INT_BLOQUE_ligne__d,
	float * normalisee__d,
	float * f, float * dif_f,
	float * y, uint Y_MAX)
{
	//	<<<(T, BLOQUES), (BLOQUE_ST)>>> //BLOQUE_ST*N_FLT==256

#define bloque blockIdx.y
#define f_dans_bloque threadIdx.x

	uint t = blockIdx.x;

	//	__partage__
	__shared__ uint ligne;
	if (threadIdx.x == 0)
		ligne = EMA_INT_BLOQUE_ligne__d[bloque];

	//	__partage__
	__shared__ float sh_x[N_FLTR];
	if (f_dans_bloque < N_FLTR)
		sh_x[f_dans_bloque] = normalisee__d[ligne*PRIXS*N_FLTR + (t0+t)*N_FLTR + f_dans_bloque];

	//	fonction de filtre
	float s = 0, d = 0;
	FOR(0, i, N_FLTR-1) {
		s += sqrtf(1 + fabs(     sh_x[i]        -   f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + i]  ));
		d += powf((1 + fabs((sh_x[i+1]-sh_x[i]) - dif_f[bloque*BLOQUE_ST*(N_FLTR-1) + f_dans_bloque*(N_FLTR-1) + i])), 2);
	};
	s += sqrtf(1 + fabs(sh_x[N_FLTR-1] - f[bloque*BLOQUE_ST*N_FLTR + f_dans_bloque*N_FLTR + N_FLTR-1]));

	//	s = a + b + c ...

	s = s/8-1;
	d = d/7-1;

	y[t*Y_MAX + bloque*BLOQUE_ST + f_dans_bloque] = 2*expf(-s*s -d*d)-1;
};

#define K__dot1d__X 16
#define K__dot1d__T  4

__global__ static
void dot1d__kerd(
	uint X,
	uint T, uint Y,
	uint Y_MAX,
	//
	uint c,
	uint * DEPART_POIDS__d,
	//
	float * x__d, float * p__d, float * y__d)
{
	//	<<<(KERD(X,16),KERD(T,4)), (16,4)>>>
	uint y = threadIdx.x + blockIdx.x * blockDim.x;
	uint t = threadIdx.y + blockIdx.y * blockDim.y;

	if ( (t<T) && (y<Y) ) {
		float s = p__d[DEPART_POIDS__d[c] + y*(X+1) + X-1+1];
		FOR(0, i, X) {
			s += x__d[t*Y_MAX + i]*p__d[DEPART_POIDS__d[c] + y*(X+1) + i];
		}
		y__d[t*Y_MAX + y] = ACTIV(s);
	};
}


static __global__ void enregistrer_les_resultats__kerd(
	float * res_d, float * y__d,
	uint Y_MAX, uint T)
{
	uint t = threadIdx.x + blockIdx.x * blockDim.x;

	if (t < T) {
		res_d[t] = y__d[t*Y_MAX + 0];
	};
};

void cuda_mdt1_mdl_f_t0t1(Mdl_t * mdl, float * res, uint t0, uint t1) {
	float * res_d;
	CONTROLE_CUDA(hipMalloc((void**)&res_d, sizeof(float)*(t1-t0)));
	CONTROLE_CUDA(hipMemset(res_d, 0, sizeof(float)*(t1-t0)));

	//	--- Allocation des r0, r1 ---
	uint T = t1 - t0;
	float * r0__d, * r1__d;
	CONTROLE_CUDA(hipMalloc((void**)&r0__d, sizeof(float) * MAX_Y * T));
	CONTROLE_CUDA(hipMalloc((void**)&r1__d, sizeof(float) * MAX_Y * T));

	//	--- Mdl_t ---

	//	--- Filtres ---
	filtre__kerd<<<dim3(T, mdl->BLOQUES), dim3(BLOQUE_ST)>>>(
		t0,
		mdl->EMA_INT_BLOQUE_ligne__d,
		normalisee__d,
		mdl->f_d, mdl->dif_f_d,
		r0__d, MAX_Y
	);
	ATTENDRE_KER_CUDA();

	//	--- Pensee Perceptronnale ---
	FOR(1, c, mdl->C) {
		dim3 grille(KER_DIV(mdl->ST[c], K__dot1d__X), KER_DIV(T, K__dot1d__T));
		dim3 noyaux(K__dot1d__X, K__dot1d__T);
		dot1d__kerd<<<grille, noyaux>>>(
			mdl->ST[c-1],
			T, mdl->ST[c],
			MAX_Y,
			//
			c,
			mdl->DEPART_POIDS__d,
			//
			(c%2==0 ? r1__d : r0__d),	//x__d
			mdl->p_d,
			(c%2==0 ? r0__d : r1__d)	//y__d
		);
		ATTENDRE_KER_CUDA();
	};

	//	--- Ecrire dans res ---
	enregistrer_les_resultats__kerd<<<dim3(KER_DIV(T,256)), dim3(256)>>>(
		res_d, ((mdl->C-1)%2==0 ? r0__d : r1__d),
		MAX_Y, T);
	ATTENDRE_KER_CUDA();
	
	CONTROLE_CUDA(hipMemcpy(
		res,
		res_d,
		sizeof(float)*(t1-t0),
		hipMemcpyDeviceToHost
	));

	//
	CONTROLE_CUDA(hipFree(res_d));
};