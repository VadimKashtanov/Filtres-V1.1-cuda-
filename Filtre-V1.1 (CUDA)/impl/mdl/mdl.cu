#include "mdl.cuh"

PAS_OPTIMISER()
Mdl_t * cree_mdl(
	uint * F_par_EMA_INT,
	uint C, uint * ST)
{
	ASSERT(ST[C-1] == 1);

	Mdl_t * mdl = (Mdl_t*)malloc(sizeof(Mdl_t));

	//
	memcpy(mdl->F_par_EMA_INT, F_par_EMA_INT, sizeof(uint) * EMA_INTS);
	mdl->FS = 0;
	FOR(0, i, EMA_INTS) mdl->FS += F_par_EMA_INT[i];
	//
	ASSERT(mdl->FS % BLOQUE_ST == 0);
	//
	mdl->BLOQUES = mdl->FS / BLOQUE_ST;
	mdl->EMA_INT_BLOQUE_ligne = (uint*)malloc(sizeof(uint) * mdl->BLOQUES);
	//
	uint k=0;
	FOR(0, i, EMA_INTS) {
		FOR(0, j, F_par_EMA_INT[i]/BLOQUE_ST) mdl->EMA_INT_BLOQUE_ligne[k++] = i;
	}

	ASSERT(mdl->FS == ST[0]);

	//
	mdl->C  = C;
	mdl->ST = cpyuint(ST, C);

	//
	mdl->FILTRES = mdl->FS * N;
	mdl->POIDS = 0;
	mdl->VARS  = mdl->FS;
	mdl->LOCDS = 0;

	//	---
	mdl->DEPART_POIDS = (uint*)malloc(sizeof(uint) * COUCHES);
	mdl->DEPART_VARS  = (uint*)malloc(sizeof(uint) * COUCHES);
	mdl->DEPART_LOCDS = (uint*)malloc(sizeof(uint) * COUCHES);
	mdl->DEPART_POIDS[0] = 0;
	mdl->DEPART_VARS [0] = 0;
	mdl->DEPART_LOCDS[0] = 0;

	//	Instructions : Pensee (dot1d)
	FOR(1, i, C) {
		ASSERT(ST[i] <= MAX_Y);

		mdl->DEPART_VARS [i] = mdl->VARS ;
		mdl->DEPART_POIDS[i] = mdl->POIDS;
		mdl->DEPART_LOCDS[i] = mdl->LOCDS;
		//
		mdl->VARS  += ST[i];
		mdl->POIDS += (ST[i-1]+1)*ST[i];
		mdl->LOCDS += ST[i];
	};

	//	======= Allocation ========
	mdl->f    = (float*)malloc(sizeof(float) * mdl->FILTRES );
	mdl->p    = (float*)malloc(sizeof(float) * mdl->POIDS   );
	mdl->y    = (float*)malloc(sizeof(float) * mdl->VARS    );
	mdl->locd = (float*)malloc(sizeof(float) * mdl->LOCDS   );

	CONTROLE_CUDA(hipMalloc((void**)&mdl->f_d,    sizeof(float) * mdl->FILTRES ));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->p_d,    sizeof(float) * mdl->POIDS   ));

	CONTROLE_CUDA(hipMalloc((void**)&mdl->dp_d, sizeof(float) * mdl->POIDS ));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->dy_d, sizeof(float) * mdl->VARS  ));

	mdl->dif_f = (float*)malloc(sizeof(float) * mdl->FS * (N-1));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->dif_f_d, sizeof(float) * mdl->FS * (N-1)));

	FOR(0, i, mdl->FS) 	  prete(mdl->f + i*N_FLTR, N_FLTR);
	FOR(0, i, mdl->POIDS) mdl->p[i] = (2*rnd()-1) * 0.5;

	//	Qlq uint pour cuda
	CONTROLE_CUDA(hipMalloc((void**)&mdl->DEPART_POIDS__d,         sizeof(uint) * mdl->C      ));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->ST__d,                   sizeof(uint) * mdl->C      ));
	CONTROLE_CUDA(hipMalloc((void**)&mdl->EMA_INT_BLOQUE_ligne__d, sizeof(uint) * mdl->BLOQUES));
	//
	CONTROLE_CUDA(hipMemcpy(mdl->DEPART_POIDS__d,         mdl->DEPART_POIDS,         sizeof(uint)*mdl->C,       hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->ST__d, 	               mdl->ST,                   sizeof(uint)*mdl->C,       hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->EMA_INT_BLOQUE_ligne__d, mdl->EMA_INT_BLOQUE_ligne, sizeof(uint)*mdl->BLOQUES, hipMemcpyHostToDevice));

	prep_mdl(mdl);

	return mdl;
};

void mdl_liberer(Mdl_t * mdl) {
	free(mdl->ST);
	free(mdl->EMA_INT_BLOQUE_ligne);
	//
	free(mdl->DEPART_POIDS);
	free(mdl->DEPART_VARS);
	free(mdl->DEPART_LOCDS);
	//
	free(mdl->dif_f);
	//
	free(mdl->f);
	free(mdl->p);
	free(mdl->y);
	free(mdl->locd);
	//
	CONTROLE_CUDA(hipFree(mdl->f_d));
	CONTROLE_CUDA(hipFree(mdl->p_d));
	CONTROLE_CUDA(hipFree(mdl->dp_d));
	CONTROLE_CUDA(hipFree(mdl->dy_d));
	CONTROLE_CUDA(hipFree(mdl->dif_f_d));
	//
	CONTROLE_CUDA(hipFree(mdl->ST__d));
	CONTROLE_CUDA(hipFree(mdl->DEPART_POIDS__d));
	CONTROLE_CUDA(hipFree(mdl->EMA_INT_BLOQUE_ligne__d));
};

void prep_mdl(Mdl_t * mdl) {
	memset(mdl->y, 0, sizeof(float) * mdl->VARS);
	memset(mdl->locd, 0, sizeof(float) * mdl->LOCDS);

	CONTROLE_CUDA(hipMemcpy(mdl->p_d, mdl->p, sizeof(float)*mdl->POIDS, hipMemcpyHostToDevice));
	CONTROLE_CUDA(hipMemcpy(mdl->f_d, mdl->f, sizeof(float)*mdl->FILTRES, hipMemcpyHostToDevice));

	CONTROLE_CUDA(hipMemset(mdl->dp_d,    0, sizeof(float) * mdl->POIDS));
	CONTROLE_CUDA(hipMemset(mdl->dy_d,    0, sizeof(float) * mdl->VARS));
	
	FOR(0, i, mdl->FS) {
		FOR(0, j, N-1) {
			mdl->dif_f[i*(N-1) + j] = mdl->f[i*N+j+1]-mdl->f[i*N+j];
		}
	}
	hipMemcpy(mdl->dif_f_d, mdl->dif_f, sizeof(float)*mdl->FS*(N-1), hipMemcpyHostToDevice);
};

void gpu_vers_cpu(Mdl_t * mdl) {
	CONTROLE_CUDA(hipMemcpy(mdl->p, mdl->p_d, sizeof(float)*mdl->POIDS, hipMemcpyDeviceToHost));
	CONTROLE_CUDA(hipMemcpy(mdl->f, mdl->f_d, sizeof(float)*mdl->FILTRES, hipMemcpyDeviceToHost));
};

void reinit_mdl(Mdl_t * mdl) {
	prep_mdl(mdl);
};