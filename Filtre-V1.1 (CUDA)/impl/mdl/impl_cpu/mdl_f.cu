#include "hip/hip_runtime.h"
#include "mdl.cuh"

static float filtre(float * x, float * dif_x, float * f, float * dif_f) {
	float s = 0, d = 0;
	FOR(0, i, N-1) {
		s += sqrtf(1 + fabs(  x[i]   -   f[i]  ));
		d += powf((1 + fabs(dif_x[i] - dif_f[i])), 2);
	};
	s += sqrtf(1 + fabs(x[N-1] - f[N-1]));

	s = s/8-1;
	d = d/7-1;

	return 2*expf(-s*s -d*d)-1;
};

static float perceptron(float * x, float * p, uint _N) {
	float s = p[_N-1+1];
	FOR(0, i, _N) s += x[i]*p[i];
	return ACTIV(s);
};

float f(Mdl_t * mdl, uint t) {
	//	--- Filtres ---
	FOR(0, b, mdl->BLOQUES) {
		FOR(0, i, BLOQUE_ST) {
			uint ligne = mdl->EMA_INT_BLOQUE_ligne[b];
			mdl->y[b*BLOQUE_ST + i] = filtre(
					normalisee + ligne*PRIXS*N_FLTR + t*N_FLTR,
				dif_normalisee + ligne*PRIXS*N_FLTR + t*N_FLTR,
				mdl->f     + b*BLOQUE_ST*N     + i*N,
				mdl->dif_f + b*BLOQUE_ST*(N-1) + i*(N-1)
			);
		};
	};
	
	//	Pensee
	FOR(1, c, mdl->C) {
		FOR(0, y, mdl->ST[c]) {
			mdl->y[mdl->DEPART_VARS[c] + y] = perceptron(
				mdl->y + mdl->DEPART_VARS[c-1],
				mdl->p + mdl->DEPART_POIDS[c] + y*(mdl->ST[c-1]+1),
				mdl->ST[c-1]
			);
		};
	};
	return mdl->y[mdl->VARS-1];
};